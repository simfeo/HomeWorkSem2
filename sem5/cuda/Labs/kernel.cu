
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <fstream>
#include <time.h>
#include <numeric>
#include <random>
#include <ctime>
#include <chrono>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i]*3 + b[i] -8;
}

int main(int argc, char** argv)
{
	// generate 2 <<30 random integers

	const int CH_BUF_LEN = 32*32;

	
	std::ifstream infileA("file_a.txt", std::ifstream::binary);
	if (!infileA.is_open())
	{
		std::cerr << "no file test_a.txt" << std::endl;
		return 1;
	}

	std::ifstream infileB("file_b.txt", std::ifstream::binary);
	if (!infileB.is_open())
	{
		std::cerr << "no file test_b.txt" << std::endl;
		return 2;
	}

	std::ofstream outfile("new.txt", std::ofstream::binary);
	if (!outfile.is_open())
	{
		std::cerr << "cannot create new.txt" << std::endl;
		return 2;
	}


	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		std::cerr<< "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?" <<std::endl;
		return 12;
	}

	auto start = std::chrono::system_clock::now();
	std::time_t start_time = std::chrono::system_clock::to_time_t(start);
#pragma warning(suppress : 4996)
	std::cout << "start at " << std::ctime(&start_time) << std::endl;

	// get size of file
	infileA.ignore(std::numeric_limits<std::streamsize>::max());
	std::streamsize sizeA = infileA.gcount();
	infileA.clear();   //  Since ignore will have set eof.
	infileA.seekg(0, std::ios_base::beg);

	infileB.ignore(std::numeric_limits<std::streamsize>::max());
	std::streamsize sizeB = infileA.gcount();
	infileB.clear();   //  Since ignore will have set eof.
	infileB.seekg(0, std::ios_base::beg);

	if (sizeA != sizeB)
	{
		std::cerr << "file_a.txt size not match file_b.txt size" << std::endl;
		return 4;
	}

	int tmpBufA[CH_BUF_LEN] = { 0 };
	int tmpBufB[CH_BUF_LEN] = { 0 };
	int tmpBufC[CH_BUF_LEN] = { 0 };


	for (int i = 0; i < sizeA; i += CH_BUF_LEN * sizeof(int))
	{
		infileA.seekg(i);
		infileB.seekg(i);
		// read content of infile
		infileA.read((char*)tmpBufA, CH_BUF_LEN * sizeof(int));
		infileA.read((char*)tmpBufB, CH_BUF_LEN * sizeof(int));

		// write to outfile

		// Add vectors in parallel.
		cudaStatus = addWithCuda(tmpBufC, tmpBufA , tmpBufC, CH_BUF_LEN);
		if (cudaStatus != hipSuccess) {
			std::cout << "addWithCuda failed!" << std::endl;;
			return 1;
		}	

		outfile.write((char*)tmpBufC, CH_BUF_LEN * sizeof(int));
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		std::cerr << "hipDeviceReset failed!" << std::endl;
		return 1;
	}

	outfile.close();
	//infileA.close();
	//infileB.close();

	auto end = std::chrono::system_clock::now();
	std::time_t end_time = std::chrono::system_clock::to_time_t(end);
	std::chrono::duration<double> elapsed_seconds = end - start;
#pragma warning(suppress : 4996)
	std::cout << "finished at " << std::ctime(&end_time) << "elapsed time: " << elapsed_seconds.count() << "s" << std::endl;

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<256, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
