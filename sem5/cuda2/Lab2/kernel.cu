#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#include "cpu_anim.h"

#define DIMX 360*2
#define DIMY 270*2

#define W DIMX //i have a large display so mul2
#define H DIMY
#define MAXX 3.1415628*2 // ������� �� ��� X - 2*PI
#define MAXY 1,5
#define MINY -1
#define DT 0.2


__global__ void kernel(unsigned char *ptr, int ticks)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float c0, c1, c2;

	c0 = abs(H / (MAXY - MINY)*(4*cos(tan((x + 0.0) / W * (MAXX + ticks * DT))) - MINY) - y);
	c1 = abs(H / (MAXY - MINY)*(4*cos(tan((x + 0.5) / W * (MAXX + ticks * DT))) - MINY) - y);
	c2 = abs(H / (MAXY - MINY)*(4*cos(tan((x - 0.5) / W * (MAXX + ticks * DT))) - MINY) - y);

	if (c0 <= 1 || c1 <= 1 || c2 <= 1)
		ptr[offset * 4 + 1] = ptr[offset * 4 + 2] = 0;
	else
		ptr[offset * 4 + 1] = ptr[offset * 4 + 2] = 255;

	ptr[offset * 4 + 0] = 255;
	ptr[offset * 4 + 3] = 255;
}

struct DataBlock
{
	unsigned char *dev_bitmap;
	CPUAnimBitmap *bitmap;
};

// ���������� ���������� ������ ����������
void cleanup(DataBlock *d)
{
	hipFree(d->dev_bitmap);
}

void generate_frame(DataBlock *d, int ticks)
{
	dim3 blocks(DIMX / 16, DIMY / 16);
	dim3 threads(16, 16);
	kernel <<<blocks, threads >>>(d->dev_bitmap, ticks);

	hipMemcpy(d->bitmap->get_ptr(),
		d->dev_bitmap,
		d->bitmap->image_size(),
		hipMemcpyDeviceToHost
	);
}

int main(void)
{
	int count = 0;

	hipGetDeviceCount(&count);

	if (!count)
	{
		std::cerr << "Not enough cuda devices =(" << std::endl;
		return 1;
	}

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	
	DataBlock data;
	CPUAnimBitmap bitmap(DIMX, DIMY, &data);
	data.bitmap = &bitmap;

	hipMalloc((void**)&data.dev_bitmap,
		bitmap.image_size()
	);
	bitmap.anim_and_exit((void(*)(void*, int))generate_frame,
		(void(*)(void*))cleanup
	);
	return 0;
}




/*
#include <GL\glut.h>

void display()
{
glClear(GL_COLOR_BUFFER_BIT);
glBegin(GL_LINES);
glColor3f(1.0, 0.0, 0.0);
glVertex2f(0.4, 0.4);
glVertex2f(0.4, 0.8);
glVertex2f(0.4, 0.8);
glVertex2f(0.8, 0.8);
glVertex2f(0.8, 0.8);
glVertex2f(0.8, 0.4);
glVertex2f(0.4, 0.4);
glVertex2f(0.8, 0.4);
glEnd();
glFlush();
}


int main(int argc, char **argv)
{
glutInit(&argc, argv);
glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);
glutInitWindowSize(240, 240);
glutInitWindowPosition(100, 740);
glutCreateWindow("First window!");
glClearColor(1.0, 1.0, 1.0, 1.0);
glMatrixMode(GL_PROJECTION);
glLoadIdentity();
glOrtho(-1.0, 1.0, -1.0, 1.0, -1.0, 1.0);
glutDisplayFunc(display);
glutMainLoop();

return 0;
}
*/